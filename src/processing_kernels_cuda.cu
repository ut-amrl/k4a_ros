#include "hip/hip_runtime.h"
// Copyright (c) 2021 Joydeep Biswas joydeepb@cs.utexas.edu

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <iostream>
#include <math.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "glog/logging.h"
// #include "eigen3/Eigen/Dense"
// #include "eigen3/Eigen/Geometry"
#include "shared/util/timer.h"
#include "processing_kernels.h"

#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/tuple.h>
#include <algorithm>

#ifndef __global__
#define __global__
#endif

#ifndef __device__
#define __device__
#endif

struct SimpleVector3f {
  float x;
  float y;
  float z;

  __device__ __host__ 
  SimpleVector3f() {}

  __device__ __host__
  SimpleVector3f(float x, float y, float z) : x(x), y(y), z(z) {}

  // SimpleVector3f& operator=(SimpleVector3f&& other) {
  //   x = other.x;
  //   y = other.y;
  //   z = other.z;
  //   return *this;
  // }

  // SimpleVector3f& operator=(const SimpleVector3f& other) {
  //   x = other.x;
  //   y = other.y;
  //   z = other.z;
  //   return *this;
  // }

  // Vector binary operators.
  #define VECTORBINARYOP(op) \
    __device__ __host__ \
    SimpleVector3f operator op(const SimpleVector3f& other) const { \
      return SimpleVector3f( \
        x op other.x, \
        y op other.y, \
        z op other.z); \
    }
  VECTORBINARYOP(+);
  VECTORBINARYOP(-);
  VECTORBINARYOP(*);
  VECTORBINARYOP(/);

  // Scalar binary operators.
  #define SCALARBINARYOP(op) \
    __device__ __host__ \
    SimpleVector3f operator op(const float& c) const { \
      return SimpleVector3f( \
        x op c, \
        y op c, \
        z op c); \
    }
  // SCALARBINARYOP(+);
  // SCALARBINARYOP(-);
  SCALARBINARYOP(*);
  SCALARBINARYOP(/);

};

struct DepthTo3D {
  __device__ __host__
  SimpleVector3f operator() (const uint16_t& depth, const SimpleVector3f& ray) {
    return translation_ + (ray * depth);
  }
  SimpleVector3f translation_;
};

thrust::device_vector<uint16_t> depth_data_;
thrust::device_vector<SimpleVector3f> rgbd_ray_lookup_;
DepthTo3D depth_to_3d_;

void InitializeTransform(const float* ray_lookups,
                         const float* translation,
                         int N) {
  const SimpleVector3f* ray_lookups_vector = 
      reinterpret_cast<const SimpleVector3f*>(ray_lookups);
  rgbd_ray_lookup_.resize(N);
  thrust::copy(ray_lookups_vector, ray_lookups_vector + N, rgbd_ray_lookup_.begin());
  depth_to_3d_.translation_.x = translation[0];
  depth_to_3d_.translation_.y = translation[1];
  depth_to_3d_.translation_.z = translation[2];
}

void DepthImageToPointCloud(const uint16_t* depth_image, 
                            int N,
                            float* point_cloud) {
  static thrust::device_vector<uint16_t> depth_image_d;
  static thrust::device_vector<SimpleVector3f> point_cloud_d;
  depth_image_d.resize(N);
  point_cloud_d.resize(N);
  thrust::copy(depth_image, depth_image + N, depth_image_d.begin());
  thrust::transform(depth_image_d.begin(),
                    depth_image_d.end(),
                    rgbd_ray_lookup_.begin(),
                    point_cloud_d.begin(),
                    depth_to_3d_);
  thrust::copy(point_cloud_d.begin(),
               point_cloud_d.end(),
               reinterpret_cast<SimpleVector3f*>(point_cloud));
}

void TestCopy(int n, const float* src, float* dest) {
  const SimpleVector3f* src_vector = 
      reinterpret_cast<const SimpleVector3f*>(src);
  SimpleVector3f* dest_vector = 
      reinterpret_cast<SimpleVector3f*>(dest);
  rgbd_ray_lookup_.resize(n);
  thrust::copy(src_vector, src_vector + n, rgbd_ray_lookup_.begin());
  thrust::copy(rgbd_ray_lookup_.begin(), rgbd_ray_lookup_.end(), dest_vector);
}

// using Eigen::Vector3f;
// using Eigen::Affine3f;

// Kernel function to add the elements of two arrays
__global__
void Add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = sinf(x[i]) + cosf(y[i]);
}

__global__
void Add2(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
    y[i] = sin(x[i]) + cos(y[i]);
}

__global__
void Add3(int n, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  y[i] = sinf(x[i]) + cosf(y[i]);
}

__device__
void MatMult(const float* m, float* v1, float* v2) {
  v2[0] = m[0] * v1[0] + m[4] * v1[1] + m[8] * v1[2] + m[12];
  v2[1] = m[1] * v1[0] + m[5] * v1[1] + m[9] * v1[2] + m[13];
  v2[2] = m[2] * v1[0] + m[6] * v1[1] + m[10] * v1[2] + m[14];
}

__global__
void Transform2(int n, const float* m, float* v1, float* v2) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int di = 3 * stride;
  int imax = 3 * n;
  for (int i = 3 * index; i < imax; i += di) {
    MatMult(m, v1 + i, v2 + i);
  }
}

void GetCapabilities() {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Blocks per multiprocessor: %d\n", 
        prop.maxBlocksPerMultiProcessor);
    printf("  Multiprocessor count: %d\n", prop.multiProcessorCount);
    
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}


// __global__
// void Transform(int n, int stride, const Affine3f tf, Vector3f* v1, Vector3f* v2) {
//   int start = (blockIdx.x * blockDim.x + threadIdx.x) * stride;
//   int end = min(n, start + stride);
//   for (int i = start; i < end; ++i) {
//     v2[i] = tf * v1[i];
//   }
// }


uint16_t* cuda_depth_image = nullptr;
float* cuda_tf = nullptr;
float* cuda_points = nullptr;
float* cuda_point_lookups = nullptr;
float* cuda_costmap = nullptr;
uint32_t* cuda_point_indices = nullptr;
int cuda_device = -1;
// Depth image size, hence point cloud size.
int N = 0;
// const float costmap_resolution = 0.05;
const int costmap_size = 100;

void InitCuda(int depth_image_size, float* point_lookups) {
  hipGetDevice(&cuda_device);
  N = depth_image_size;

  CHECK_EQ(hipMalloc(&cuda_tf, 12 * sizeof(float)), hipSuccess);
  CHECK_EQ(hipMalloc(&cuda_depth_image, N * sizeof(uint16_t)), hipSuccess);
  CHECK_EQ(hipMalloc(&cuda_points, 3 * N * sizeof(float)), hipSuccess);
  CHECK_EQ(hipMalloc(&cuda_point_lookups, 3 * N * sizeof(float)),
      hipSuccess);
  CHECK_EQ(hipMalloc(&cuda_costmap, costmap_size * costmap_size * 
      sizeof(float)), hipSuccess);
  CHECK_EQ(hipMalloc(&cuda_point_indices, N * sizeof(uint32_t)), hipSuccess);

  // Copy the lookups to GPU memory.
  CHECK_EQ(hipMemcpy(cuda_point_lookups, point_lookups, 3 * N * sizeof(float), hipMemcpyHostToDevice), hipSuccess);

}

void DepthToCostmap(float* depth_image, float* tf, float* costmap) {
  
  // Inputs: depth image, point lookups, tf
  // Output: costmap
  
  // Copy depth image to CUDA
  CHECK_EQ(hipMemcpy(cuda_depth_image, depth_image, N * sizeof(float), 
      hipMemcpyHostToDevice), hipSuccess);
  // Copy tf to CUDA
  CHECK_EQ(hipMemcpy(cuda_tf, tf, 12 * sizeof(float), hipMemcpyHostToDevice), 
      hipSuccess);

  // Run depth to point cloud & indexer kernel.
  // Run costmap projection kernel.
  // Run costmap propagation kernel.

  // Copy costmap to CPU.
  CHECK_EQ(hipMemcpy(costmap, cuda_costmap, costmap_size * costmap_size * 
      sizeof(float), hipMemcpyDeviceToHost), hipSuccess);
  // Return costmap.
}

void TestCuda() {
}

// void TestCuda() {
//   GetCapabilities();
//   int N = 1920*1080;
//   Vector3f* v1 = nullptr;
//   Vector3f* v2 = nullptr;
//   Vector3f* v3 = new Vector3f[N];
//   hipMallocManaged(&v1, N*sizeof(Vector3f));
//   hipMallocManaged(&v2, N*sizeof(Vector3f));
//   hipMemPrefetchAsync(v1, N*sizeof(Vector3f), hipCpuDeviceId, NULL);
//   hipMemPrefetchAsync(v2, N*sizeof(Vector3f), hipCpuDeviceId, NULL);

//   Affine3f tf = Eigen::Translation3f(Vector3f(1, 0, 0)) * 
//       Eigen::AngleAxisf(1.0, Vector3f(0, 1, 0));

//   for (int i = 0; i < N; ++i) {
//     v1[i] = Vector3f(1, 2, 3);
//   }

//   {
//     FunctionTimer ft("CPU");
//     for (int i = 0; i < N; ++i) {
//       v3[i] = tf * v1[i];
//     }
//   }

//   int device = -1;
//   hipGetDevice(&device);
//   hipMemPrefetchAsync(v1, N*sizeof(Vector3f), device, NULL);
//   hipMemPrefetchAsync(v2, N*sizeof(Vector3f), device, NULL);
//   {
//     FunctionTimer ft("GPU");
//     int stride = 128;
//     int blockSize = 1024;
//     int numBlocks = ((N + stride - 1) / stride + blockSize - 1) / blockSize;
//     printf("%d blocks, %d threads\n", numBlocks, blockSize);
//     if (true) {
//       Transform<<<numBlocks, blockSize>>>(N, stride, tf, v1, v2);
//     } else {
//       Eigen::Matrix4f m = tf.matrix();
//       Transform2<<<numBlocks, blockSize>>>(
//           N, 
//           m.data(), 
//           reinterpret_cast<float*>(v1), 
//           reinterpret_cast<float*>(v2));
//     }
//     // Wait for GPU to finish before accessing on host
//     hipDeviceSynchronize();
//   }
//   hipMemPrefetchAsync(v1, N*sizeof(Vector3f), hipCpuDeviceId, NULL);
//   hipMemPrefetchAsync(v2, N*sizeof(Vector3f), hipCpuDeviceId, NULL);
//   float max_error = 0;
//   for (int i = 0; i < N; ++i) {
//     max_error = max(max_error, (v3[i] - v2[i]).norm());
//   }
//   printf("Max error: %f\n", max_error);

//   hipFree(v1);
//   hipFree(v2);
//   delete[] v3;
// }

// void TestCuda() {
//   GetCapabilities();
//   int N = 1<<26;
//   float *x, *y;
//   float *z = new float[N];

//   // Allocate Unified Memory – accessible from CPU or GPU
//   hipMallocManaged(&x, N*sizeof(float));
//   hipMallocManaged(&y, N*sizeof(float));

//   // initialize x and y arrays on the host
//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//     z[i] = sin(x[i]) + cos(y[i]);
//   }

//   {
//     FunctionTimer ft("CPU");
//     for (int i = 0; i < N; i++) {
//       z[i] = sin(x[i]) + cos(y[i]);
//     }
//   }
//   // Run kernel on 1M elements on the GPU
//   if (true) {
//     FunctionTimer ft("GPU");
//     int blockSize = 1024;
//     int numBlocks = (N + 1) / blockSize;
//     printf("%d blocks, %d threads\n", numBlocks, blockSize);
//     Add3<<<numBlocks, blockSize>>>(N, x, y);
//     // Add2<<<1, 256>>>(N, x, y);

//     // Wait for GPU to finish before accessing on host
//     hipDeviceSynchronize();
//   } else {
//     FunctionTimer ft("GPU");
//     int blockSize = 1024;
//     int numBlocks = (N + blockSize - 1) / blockSize;
//     printf("%d blocks, %d threads\n", numBlocks, blockSize);
//     Add<<<numBlocks, blockSize>>>(N, x, y);
//     // Add2<<<1, 256>>>(N, x, y);

//     // Wait for GPU to finish before accessing on host
//     hipDeviceSynchronize();
//   }

//   // Check for errors (all values should be 3.0f)
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++) {
//     maxError = fmax(maxError, fabs(y[i]-z[i]));
//   }
//   std::cout << "Max error: " << maxError << std::endl;

//   // Free memory
//   hipFree(x);
//   hipFree(y);
//   delete z;
// }
