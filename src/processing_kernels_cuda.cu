#include "hip/hip_runtime.h"
// Copyright (c) 2021 Joydeep Biswas joydeepb@cs.utexas.edu

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <iostream>
#include <math.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "glog/logging.h"
// #include "eigen3/Eigen/Dense"
// #include "eigen3/Eigen/Geometry"
#include "shared/util/timer.h"
#include "processing_kernels.h"

#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/tuple.h>
#include <algorithm>

#ifndef __global__
#define __global__
#endif

#ifndef __device__
#define __device__
#endif


// Anonymous namespace to prevent collisions with external use.
namespace {
struct SimpleVector3f {
  float x;
  float y;
  float z;

  __device__ __host__ 
  SimpleVector3f() {}

  __device__ __host__
  SimpleVector3f(float x, float y, float z) : x(x), y(y), z(z) {}

  // SimpleVector3f& operator=(SimpleVector3f&& other) {
  //   x = other.x;
  //   y = other.y;
  //   z = other.z;
  //   return *this;
  // }

  // SimpleVector3f& operator=(const SimpleVector3f& other) {
  //   x = other.x;
  //   y = other.y;
  //   z = other.z;
  //   return *this;
  // }

  // Vector binary operators.
  #define VECTORBINARYOP(op) \
    __device__ __host__ \
    SimpleVector3f operator op(const SimpleVector3f& other) const { \
      return SimpleVector3f( \
        x op other.x, \
        y op other.y, \
        z op other.z); \
    }
  VECTORBINARYOP(+);
  VECTORBINARYOP(-);
  VECTORBINARYOP(*);
  VECTORBINARYOP(/);

  // Scalar binary operators.
  #define SCALARBINARYOP(op) \
    __device__ __host__ \
    SimpleVector3f operator op(const float& c) const { \
      return SimpleVector3f( \
        x op c, \
        y op c, \
        z op c); \
    }
  // SCALARBINARYOP(+);
  // SCALARBINARYOP(-);
  SCALARBINARYOP(*);
  SCALARBINARYOP(/);

};

struct DepthTo3D {
  __device__ __host__
  SimpleVector3f operator() (const uint16_t& depth, const SimpleVector3f& ray) {
    return translation_ + (ray * depth);
  }
  SimpleVector3f translation_;
};

thrust::device_vector<uint16_t> depth_data_;
thrust::device_vector<SimpleVector3f> rgbd_ray_lookup_;
DepthTo3D depth_to_3d_;

}  // namespace

namespace processing_kernels {
void InitializeTransform(const float* ray_lookups,
                         const float* translation,
                         int N) {
  const SimpleVector3f* ray_lookups_vector = 
      reinterpret_cast<const SimpleVector3f*>(ray_lookups);
  rgbd_ray_lookup_.resize(N);
  thrust::copy(ray_lookups_vector, ray_lookups_vector + N, rgbd_ray_lookup_.begin());
  depth_to_3d_.translation_.x = translation[0];
  depth_to_3d_.translation_.y = translation[1];
  depth_to_3d_.translation_.z = translation[2];
}

void DepthImageToPointCloud(const uint16_t* depth_image, 
                            int N,
                            float* point_cloud) {
  static thrust::device_vector<uint16_t> depth_image_d;
  static thrust::device_vector<SimpleVector3f> point_cloud_d;
  depth_image_d.resize(N);
  point_cloud_d.resize(N);
  thrust::copy(depth_image, depth_image + N, depth_image_d.begin());
  thrust::transform(depth_image_d.begin(),
                    depth_image_d.end(),
                    rgbd_ray_lookup_.begin(),
                    point_cloud_d.begin(),
                    depth_to_3d_);
  thrust::copy(point_cloud_d.begin(),
               point_cloud_d.end(),
               reinterpret_cast<SimpleVector3f*>(point_cloud));
}

void TestCopy(int n, const float* src_h, float* dest_h) {
  // Allocate device memory.
  thrust::device_vector<float> tmp_d(n);
  // Host to cuda.
  thrust::copy(src_h, src_h + n, tmp_d.begin());
  // Cuda to host.
  thrust::copy(tmp_d.begin(), tmp_d.end(), dest_h);
}

void GetCudaCapabilities() {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Threads per block: %d\n", prop.maxThreadsPerBlock);
    //printf("  Blocks per multiprocessor: %d\n", 
    //    prop.maxBlocksPerMultiProcessor);
    printf("  Multiprocessor count: %d\n", prop.multiProcessorCount);
    
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
}  // namespace processing_kernels
